#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, Po, , );
rtDeclareVariable(float3, N, , );
rtDeclareVariable(float, R, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx) {
	float3 Lo = ray.origin;
	float3 L = ray.direction;
	float t =  dot(Po-Lo, N) / dot(L, N);
	if (rtPotentialIntersection(t)) {
		geometric_normal = shading_normal = N;
		rtReportIntersection(0);
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (length(N) > 0 && R > 0) {
		float3 rad = make_float3(R, 0.0f, R);
		aabb->m_max = Po + rad;
		aabb->m_min = Po - rad;
	} else {
		aabb->invalidate();
	}
}