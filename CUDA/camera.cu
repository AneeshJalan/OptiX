#include "hip/hip_runtime.h"

#include <optix_world.h>
#include "Common.h"

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, objects, , );

rtBuffer<uchar4, 2> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

RT_PROGRAM void raygen() {
	float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.0f - 1.0f;
	float3 dir = normalize(d.x*U + d.y*V + W);
  
	optix::Ray ray = optix::make_Ray(eye, dir, RADIANCE_RAY, DEFAULT_MIN, RT_DEFAULT_MAX);

	PerRayData_radiance data;
	data.importance = 1.0f;
	data.depth = 0;

	rtTrace(objects, ray, data);

	output_buffer[launch_index] = optix::make_uchar4(static_cast<unsigned char>(__saturatef(data.result.z)*255.99f),  /* B */
													 static_cast<unsigned char>(__saturatef(data.result.y)*255.99f),  /* G */
													 static_cast<unsigned char>(__saturatef(data.result.x)*255.99f),  /* R */
													 255u);
}

rtDeclareVariable(float3, exception_color, , );

RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = optix::make_uchar4(static_cast<unsigned char>(__saturatef(exception_color.z)*255.99f),  /* B */
													 static_cast<unsigned char>(__saturatef(exception_color.y)*255.99f),  /* G */
													 static_cast<unsigned char>(__saturatef(exception_color.x)*255.99f),  /* R */
													 255u);
}

rtDeclareVariable(float3, background, , );
rtDeclareVariable(PerRayData_radiance, data, rtPayload, );

RT_PROGRAM void miss() {
	data.result = background;
}