#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>
#include "Common.h"

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, objects, , );

rtBuffer<uchar4, 2> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

__device__ float correct(float color) {
	return clamp(color, 0.0f, 1.0f);
}

__device__ uchar4 convertColor(float3 clr) {
	return make_uchar4(static_cast<unsigned char>(correct(clr.z) * 255.99f),  /* B */
					   static_cast<unsigned char>(correct(clr.y) * 255.99f),  /* G */
					   static_cast<unsigned char>(correct(clr.x) * 255.99f),  /* R */
					   255u);
}

#define NUM_SAMPLES 4

RT_PROGRAM void raygen() {
	//Rotated-Grid AA
	float2 offset[] = {
	//	make_float2(0.0f, 0.0f),
		make_float2(1/8.0f, 3/8.0f),
		make_float2(3/8.0f, 1/8.0f),
		make_float2(5/8.0f, 7/8.0f),
		make_float2(7/8.0f, 5/8.0f)
	};

	float3 color = make_float3(0.0f, 0.0f, 0.0f);

	for (int i = 0; i < NUM_SAMPLES; i++) {
		float2 d = (make_float2(launch_index) + offset[i]) / make_float2(launch_dim) * 2.0f - 1.0f;
		float3 dir = normalize(d.x*U + d.y*V + W);
  
		optix::Ray ray = optix::make_Ray(eye, dir, RADIANCE_RAY, DEFAULT_MIN, 1e8);

		PerRayData_radiance data;
		data.importance = 1.0f;
		data.depth = 0;

		rtTrace(objects, ray, data);

		color += data.result;
	}
	
	output_buffer[launch_index] = convertColor(color / NUM_SAMPLES);
}

rtDeclareVariable(float3, exception_color, , );

RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = convertColor(exception_color);
}

//rtDeclareVariable(float3, background, , );
rtDeclareVariable(PerRayData_radiance, data, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtTextureSampler<float4, 2> envmap;

RT_PROGRAM void miss() {
//	data.result = background;
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	data.result = make_float3(tex2D(envmap, u, v));
}