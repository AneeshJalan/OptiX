#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
//#include "algebra.h"

using namespace optix;

rtDeclareVariable(float2, radii, , );	//major, minor

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(Ray, ray, rtCurrentRay, );
/*
RT_PROGRAM void intersect(int primIdx) {
	float3 O = ray.origin;
	float3 D = ray.direction;
	double ab = dot(O, D);
	double aa = dot(O, O);

	double R2 = radii.x*radii.x;
	double r2 = radii.y*radii.y;
	double K = aa - r2 - R2;
	double constants[5], roots[4];
	constants[4] = 1.0f;
	constants[3] = 4*ab;
	constants[2] = 2*(2*ab*ab + K + 2*R2*D.z*D.z);
	constants[1] = 4*(K*ab + 2*R2*O.z*D.z);
	constants[0] = K*K + 4*R2*(O.z*O.z - r2);
	int nroots = solveQuartic(constants, roots);

	int intersection = 0;
	double min = ray.tmax;
	while(nroots--) {
		double t = roots[nroots];
		//double x = O.x + t*D.x;
		//double y = O.y + t*D.y;
		//double l = radii.x*(M_PI/2 - atan2(y,x));
		//if (l >= 0) {
			if (t < min) {
				if (rtPotentialIntersection(t)) {
					min = t;
					intersection = 1;
				}
			}
		//}
	}
	if (intersection) {
		float3 pos = O + min*D;
		float3 rel = pos; rel.z = 0;
		rel = radii.x * normalize(rel);
		shading_normal = geometric_normal = pos - rel;
        rtReportIntersection(0);
	}
}
*/
RT_PROGRAM void intersect(int primIdx) {
	float3 ro = ray.origin;
	float3 rd = ray.direction;

	double Ra2 = radii.x*radii.x;
	double ra2 = radii.y*radii.y;
	
	double m = dot(ro,ro);
	double n = dot(ro,rd);
		
	double k = (m - ra2 - Ra2)/2.0;
	double a = n;
	double b = n*n + Ra2*rd.z*rd.z + k;
	double c = k*n + Ra2*ro.z*rd.z;
	double d = k*k + Ra2*ro.z*ro.z - Ra2*ra2;
	
    //----------------------------------

	double p = -3.0*a*a     + 2.0*b;
	double q =  2.0*a*a*a   - 2.0*a*b   + 2.0*c;
	double r = -3.0*a*a*a*a + 4.0*a*a*b - 8.0*a*c + 4.0*d;
	p /= 3.0;
	r /= 3.0;
	double Q = p*p + r;
	double R = 3.0*r*p - p*p*p - q*q;
	
	double h = R*R - Q*Q*Q;
	double z = 0.0;
	if( h < 0.0 )
	{
		double sQ = sqrt(Q);
		z = 2.0*sQ*cos( acos(R/(sQ*Q)) / 3.0 );
	}
	else
	{
		double sQ = powf( sqrt(h) + abs(R), 1.0/3.0 );
		z = abs( sQ + Q/sQ );
		if (R < 0) z = -z;
	}
	
	z = p - z;
	
    //----------------------------------
	
	double d1 = z   - 3.0*p;
	double d2 = z*z - 3.0*r;

	if( abs(d1)<1.0e-3 )
	{
		if( d2<0.0 ) return;
		d2 = sqrt(d2);
	}
	else
	{
		if( d1<0.0 ) return;
		d1 = sqrt( d1/2.0 );
		d2 = q/d1;
	}

    //----------------------------------
	
	double result = 1e10;

	h = d1*d1 - z + d2;
	if( h>0.0 )
	{
		h = sqrt(h);
		double t1 = -d1 - h - a;
		double t2 = -d1 + h - a;
			 if( t1>1.0e-3 ) result=t1;
		else if( t2>1.0e-3 ) result=t2;
	}

	h = d1*d1 - z - d2;
	if( h>0.0 )
	{
		h = sqrt(h);
		double t1 = d1 - h - a;
		double t2 = d1 + h - a;
		     if( t1>1.0e-3 ) result=fminf(result,t1);
		else if( t2>1.0e-3 ) result=fminf(result,t2);
	}

	if (rtPotentialIntersection(result)) {
		float3 pos = ro + result*rd;
		float3 rel = pos; rel.z = 0;
		rel = radii.x * normalize(rel);
		shading_normal = geometric_normal = pos - rel;
        rtReportIntersection(0);
	}
}

RT_PROGRAM void bounds(int, double result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (radii.x > 0 || radii.y > 0) {
		aabb->m_max = make_float3(radii.x + radii.y, radii.x + radii.y, radii.y);
		aabb->m_min = -aabb->m_max;
	} else {
		aabb->invalidate();
	}
}