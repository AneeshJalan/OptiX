#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
//#include "algebra.h"

using namespace optix;

rtDeclareVariable(float2, radii, , );	//major, minor

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(Ray, ray, rtCurrentRay, );
/*
RT_PROGRAM void intersect(int primIdx) {
	float3 O = ray.origin;
	float3 D = ray.direction;
	float ab = dot(O, D);
	float aa = dot(O, O);

	float R2 = radii.x*radii.x;
	float r2 = radii.y*radii.y;
	float K = aa - r2 - R2;
	float constants[5], roots[4];
	constants[4] = 1.0f;
	constants[3] = 4*ab;
	constants[2] = 2*(2*ab*ab + K + 2*R2*D.z*D.z);
	constants[1] = 4*(K*ab + 2*R2*O.z*D.z);
	constants[0] = K*K + 4*R2*(O.z*O.z - r2);
	int nroots = solveQuartic(constants, roots);

	int intersection = 0;
	float min = ray.tmax;
	while(nroots--) {
		float t = roots[nroots];
		//float x = O.x + t*D.x;
		//float y = O.y + t*D.y;
		//float l = radii.x*(M_PI/2 - atan2(y,x));
		//if (l >= 0) {
			if (t < min) {
				if (rtPotentialIntersection(t)) {
					min = t;
					intersection = 1;
				}
			}
		//}
	}
	if (intersection) {
		float3 pos = O + min*D;
		float3 rel = pos; rel.z = 0;
		rel = radii.x * normalize(rel);
		shading_normal = geometric_normal = pos - rel;
        rtReportIntersection(0);
	}
}
*/
RT_PROGRAM void intersect(int primIdx) {
	float3 ro = ray.origin;
	float3 rd = ray.direction;

	float Ra2 = radii.x*radii.x;
	float ra2 = radii.y*radii.y;
	
	float m = dot(ro,ro);
	float n = dot(ro,rd);
		
	float k = (m - ra2 - Ra2)/2.0;
	float a = n;
	float b = n*n + Ra2*rd.z*rd.z + k;
	float c = k*n + Ra2*ro.z*rd.z;
	float d = k*k + Ra2*ro.z*ro.z - Ra2*ra2;
	
    //----------------------------------

	float p = -3.0*a*a     + 2.0*b;
	float q =  2.0*a*a*a   - 2.0*a*b   + 2.0*c;
	float r = -3.0*a*a*a*a + 4.0*a*a*b - 8.0*a*c + 4.0*d;
	p /= 3.0;
	r /= 3.0;
	float Q = p*p + r;
	float R = 3.0*r*p - p*p*p - q*q;
	
	float h = R*R - Q*Q*Q;
	float z = 0.0;
	if( h < 0.0 )
	{
		float sQ = sqrt(Q);
		z = 2.0*sQ*cos( acos(R/(sQ*Q)) / 3.0 );
	}
	else
	{
		float sQ = powf( sqrt(h) + abs(R), 1.0/3.0 );
		z = abs( sQ + Q/sQ );
		if (R < 0) z = -z;
	}
	
	z = p - z;
	
    //----------------------------------
	
	float d1 = z   - 3.0*p;
	float d2 = z*z - 3.0*r;

	if( abs(d1)<1.0e-4 )
	{
		if( d2<0.0 ) return;
		d2 = sqrt(d2);
	}
	else
	{
		if( d1<0.0 ) return;
		d1 = sqrt( d1/2.0 );
		d2 = q/d1;
	}

    //----------------------------------
	
	float result = 1e10;

	h = d1*d1 - z + d2;
	if( h>0.0 )
	{
		h = sqrt(h);
		float t1 = -d1 - h - a;
		float t2 = -d1 + h - a;
			 if( t1>1.0e-3f ) result=t1;
		else if( t2>1.0e-3f ) result=t2;
	}

	h = d1*d1 - z - d2;
	if( h>0.0 )
	{
		h = sqrt(h);
		float t1 = d1 - h - a;
		float t2 = d1 + h - a;
		     if( t1>1.0e-3f ) result=fminf(result,t1);
		else if( t2>1.0e-3f ) result=fminf(result,t2);
	}

	if (rtPotentialIntersection(result)) {
		float3 pos = ro + result*rd;
		float3 rel = pos; rel.z = 0;
		rel = radii.x * normalize(rel);
		shading_normal = geometric_normal = pos - rel;
        rtReportIntersection(0);
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (radii.x > 0 || radii.y > 0) {
		aabb->m_max = make_float3(radii.x + radii.y, radii.x + radii.y, radii.y);
		aabb->m_min = -aabb->m_max;
	} else {
		aabb->invalidate();
	}
}