#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "algebra.h"

using namespace optix;

rtDeclareVariable(float2, radii, , );	//major, minor

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx) {
	float3 O = ray.origin;
	float3 D = ray.direction;
	float ab = dot(O, D);
	float aa = dot(O, O);

	float R2 = radii.x*radii.x;
	float r2 = radii.y*radii.y;
	float K = aa - r2 - R2;
	float constants[5], roots[4];
	constants[4] = 1.0f;
	constants[3] = 4*ab;
	constants[2] = 2*(2*ab*ab + K + 2*R2*D.z*D.z);
	constants[1] = 4*(K*ab + 2*R2*O.z*D.z);
	constants[0] = K*K + 4*R2*(O.z*O.z - r2);
	int nroots = solveQuartic(constants, roots);

	int intersection = 0;
	float min = ray.tmax;
	while(nroots--) {
		float t = roots[nroots];
		//float x = O.x + t*D.x;
		//float y = O.y + t*D.y;
		//float l = radii.x*(M_PI/2 - atan2(y,x));
		//if (/*l <= vlength && */l >= 0) {
			if (t < min) {
				if (rtPotentialIntersection(t)) {
					min = t;
					intersection = 1;
				}
			}
		//}
	}
	if (intersection) {
		float3 pos = O + min*D;
		float3 rel = pos; rel.y = 0;
		rel = radii.x * normalize(rel);
		shading_normal = geometric_normal = pos - rel;
        rtReportIntersection(0);
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (radii.x > 0 || radii.y > 0) {
		aabb->m_max = make_float3(radii.x + radii.y, radii.x + radii.y, radii.x + radii.y);
		aabb->m_max = aabb->m_max * aabb->m_max;
		aabb->m_min = -aabb->m_max;
	} else {
		aabb->invalidate();
	}
}