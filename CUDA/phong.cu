#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "Common.h"

using namespace optix;

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, ref, , );
rtDeclareVariable(float, e, , );
rtDeclareVariable(float, opacity, , );
rtDeclareVariable(float, index, , );
rtDeclareVariable(int, mode, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, dist, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, radianceData, rtPayload, );
rtDeclareVariable(rtObject, objects, , );
rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void radiance() {
	float3 world_snormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_gnormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	float3 normal = faceforward(world_snormal, -ray.direction, world_gnormal);
	float3 location = ray.origin + dist * ray.direction;
	
	if (mode == 0) {
		float3 result = Ka;

		unsigned int num_lights = lights.size();
		for(int i = 0; i < num_lights; ++i) {
			BasicLight light = lights[i];
			float distance = optix::length(light.pos - location);
			float3 L = optix::normalize(light.pos - location);
			float dot = optix::dot(normal, L);

			// cast shadow ray
			float3 attenuation = make_float3(static_cast<float>(dot > 0.0f));
			if (dot > 0.0f && light.casts_shadow) {
				PerRayData_shadow data;
				data.attenuation = make_float3(1.0f);
				optix::Ray shadow_ray = optix::make_Ray(location, L, SHADOW_RAY, DEFAULT_MIN, distance);
				rtTrace(objects, shadow_ray, data);
				//radianceData.result = data.attenuation;
				//return;
				attenuation = data.attenuation;
			}
			//radianceData.result = make_float3(0.0f);
			//return;
			//attenuation = make_float3(1.0f);

			// If not completely shadowed, light the hit point
			if (fmaxf(attenuation) > 0.0f) {
				float3 clr = light.color * attenuation;

				result += Kd * dot * clr;

				float3 H = optix::normalize(L - ray.direction);
				float dot2 = optix::dot(normal, H);
				if(dot2 > 0.0f) {
					float power = pow(dot2, e);
					result += Ks * power * clr;
				}
			}
		}

		//Reflection
		if(fmaxf(ref) > 0.0f) {
			// ray tree attenuation
			PerRayData_radiance data;             
			data.importance = radianceData.importance * optix::luminance(ref);
			data.depth = radianceData.depth + 1;

			// reflection ray
			if(data.importance >= 0.01f && data.depth <= MAX_DEPTH) {
				float3 R = optix::reflect(ray.direction, normal);
				optix::Ray reflection = optix::make_Ray(location, R, RADIANCE_RAY, DEFAULT_MIN, RT_DEFAULT_MAX);
				rtTrace(objects, reflection, data);
				result += ref * data.result;
			}
		}

		float refracted = 1.0f - opacity;
		//Refraction
	//	if (refracted > 0.0f) {
			//TODO improve this to use fresnel function to balance reflection/refraction amounts
			PerRayData_radiance data;
			data.importance = radianceData.importance * refracted;
			data.depth = radianceData.depth + 1;

			if (data.importance >= 0.1f && data.depth <= MAX_DEPTH) {
				//calculation
				float3 R;
				if (optix::refract(R, optix::normalize(ray.direction), normal, index)) {
					optix::Ray refraction = optix::make_Ray(location, R, RADIANCE_RAY, DEFAULT_MIN, RT_DEFAULT_MAX);
					rtTrace(objects, refraction, data);
					result = (opacity * result) + (refracted * data.result);
				}
				//TODO total interal reflection
			}
	//	}
		
		// pass the color back up the tree
		radianceData.result = result;
	} else if (mode == 1) {	
		radianceData.result = 0.5f * (normal + make_float3(1.0f));
	} else if (mode == 2) {
		radianceData.result = 0.33333f * (location + make_float3(1.5f));
	}	
}

rtDeclareVariable(PerRayData_shadow, shadowData, rtPayload, );

RT_PROGRAM void shadow() {
	shadowData.attenuation = optix::make_float3(0.0f);
	rtTerminateRay();
}