#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float, radius, , );
rtDeclareVariable(float3, position, , );

rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, n, attribute normal, );

RT_PROGRAM void intersect(int primIdx) {
	float3 O = ray.origin;
	float3 D = ray.direction;
	float3 C = position;

	float a = dot(D, D);
	float b = 2.0f * dot(D, O - C);
	float c = dot(O - C, O - C) - (radius * radius);

	float d = (b*b) - (4.0f*a*c);
	if (d >= 0) {
		float result = ray.tmax;
		d = sqrtf(d);
		a *= 2;
		float t = (-b + d) / a;
		if (t > 1.0e-4f) result = t;
		t = (-b - d) / a;
		if (t > 1.0e-4f) result = fminf(result, t);
		if (rtPotentialIntersection(result)) {
			float3 pos = O + result*D;
			n = normalize(pos - C);
			// TODO: pass other attributes to shader
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (radius > 0) {
		float3 R = make_float3(radius);
		aabb->m_max = position + R;
		aabb->m_min = position - R;
	} else {
		aabb->invalidate();
	}
}