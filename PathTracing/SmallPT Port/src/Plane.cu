#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, Po, , );
rtDeclareVariable(float3, N, , );
rtDeclareVariable(float3, R, , );
rtDeclareVariable(int, objID, , );

rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, n, attribute normal, );
rtDeclareVariable(int, id, attribute ID, );

RT_PROGRAM void intersect(int primIdx) {
	float3 Lo = ray.origin;
	float3 L = ray.direction;
	float t = dot(Po - Lo, N) / dot(L, N);
	if (rtPotentialIntersection(t)) {
		n = N;
		id = objID;
		rtReportIntersection(0);
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (length(N) > 0 && length(R) > 0) {
		aabb->m_max = Po + R;
		aabb->m_min = Po - R;
		return;
	}
	aabb->invalidate();
}