#include "hip/hip_runtime.h"
#include "RayData.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#define M_PI 3.141592653589793

using namespace optix;

rtDeclareVariable(float3, f, , );
rtDeclareVariable(float3, e, , );

rtDeclareVariable(Ray, r, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RayData, data, rtPayload, );
rtDeclareVariable(rtObject, objects, , );
rtDeclareVariable(int, maxDepth, , );

rtDeclareVariable(float3, n, attribute normal, );

static __device__ __inline__ float3 radiance(float3 o, float3 d, int depth) {
	RayData payload;
	payload.depth = depth;
	payload.seed = data.seed;
	optix::Ray ray = optix::make_Ray(o, d, 0, DEFAULT_MIN, DEFAULT_MAX);
	rtTrace(objects, ray, payload);
	return payload.result;
}

RT_PROGRAM void diffuse() {		// Ideal diffuse reflection
	if (data.depth >= maxDepth) {
		data.result = e;
		return;
	}
	float3 pos = r.origin + r.direction*t;
	float3 nl = dot(n, r.direction) < 0 ? n : -n;
	float p = fmaxf(f); // max reflectance

	// Possibly optional line?
//	if (++depth>5) if (randFloat()<p) f = f*(1 / p); else return obj.e;
	
	float r1 = 2 * M_PI*randFloat(data.seed);
	float r2 = randFloat(data.seed), r2s = sqrt(r2);
	float3 w = nl;
	float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0.0f, 1.0f, 0.0f) : make_float3(1.0f, 0.0f, 0.0f)), w));
	float3 v = cross(w, u);
	float3 d = normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrt(1 - r2));
	data.result = e + (f * radiance(pos, d, data.depth + 1));
}

RT_PROGRAM void specular() {	// Ideal specular reflection
	if (data.depth > maxDepth) {
		data.result = e;
		return;
	}
	float3 pos = r.origin + r.direction*t;
	float3 nl = dot(n, r.direction) < 0 ? n : -n;
	float p = fmaxf(f); // max reflectance

	// Possibly optional line?
	//	if (++depth>5) if (randFloat()<p) f = f*(1 / p); else return obj.e;

	data.result = e + (f * radiance(pos, r.direction - (n * 2 * dot(n, r.direction)), data.depth + 1));
}

RT_PROGRAM void refractive() {	// Ideal dielectric REFRACTION
	if (data.depth > maxDepth) {
		data.result = e;
		return;
	}
	float3 pos = r.origin + r.direction*t;
	float3 nl = dot(n, r.direction) < 0 ? n : -n;
	float p = fmaxf(f); // max reflectance

	// Possibly optional line?
	//	if (++depth>5) if (randFloat()<p) f = f*(1 / p); else return obj.e;

	float3 reflDir = r.direction - (n * 2 * dot(n, r.direction));
	bool into = dot(n, nl) > 0;
	float nc = 1, nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = dot(r.direction, nl), cos2t;
	if ((cos2t = 1 - nnt*nnt*(1 - ddn*ddn)) < 0) {	// Total internal reflection
		data.result = e + (f * radiance(pos, reflDir, data.depth + 1));
	}
	float3 tdir = normalize(r.direction*nnt - n*((into ? 1 : -1)*(ddn*nnt + sqrt(cos2t))));
	float a = nt - nc, b = nt + nc, R0 = a*a / (b*b), c = 1 - (into ? -ddn : dot(tdir, n));
	float Re = R0 + (1 - R0)*c*c*c*c*c, Tr = 1 - Re, P = .25 + .5*Re, RP = Re / P, TP = Tr / (1 - P);
	data.result = e + (f * (data.depth > 2 ? (randFloat(data.seed) < P ?   // Russian roulette
		radiance(pos, reflDir, data.depth + 1)*RP : radiance(pos, tdir, data.depth + 1)*TP) :
		radiance(pos, reflDir, data.depth + 1)*Re + radiance(pos, tdir, data.depth + 1)*Tr));
}