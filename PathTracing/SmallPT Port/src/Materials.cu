#include "hip/hip_runtime.h"
#include "RayData.h"
#include "SphereLight.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#define M_PI 3.141592653589793

using namespace optix;

rtBuffer<SphereLight> lights;
rtDeclareVariable(float3, f, , );
rtDeclareVariable(float3, e, , );

rtDeclareVariable(Ray, r, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RayData, data, rtPayload, );
rtDeclareVariable(rtObject, objects, , );
rtDeclareVariable(int, maxDepth, , );

rtDeclareVariable(float3, n, attribute normal, );
rtDeclareVariable(int, id, attribute ID, );

static __device__ __inline__ float3 radiance(float3 o, float3 d, int depth, int E = 1) {
	RayData payload;
	payload.E = E;
	payload.depth = depth;
	payload.seed = data.seed;
	optix::Ray ray = optix::make_Ray(o, d, 0, DEFAULT_MIN, DEFAULT_MAX);
	rtTrace(objects, ray, payload);
	return payload.result;
}

RT_PROGRAM void diffuse() {		// Ideal diffuse reflection
	data.id = id;
	if (data.depth >= maxDepth) {
		data.result = e * data.E;
		return;
	}
	float3 pos = r.origin + r.direction*t;
	float3 nl = dot(n, r.direction) < 0 ? n : -n;
	float p = fmaxf(f); // max reflectance

	float r1 = 2 * M_PI*randFloat(data.seed);
	float r2 = randFloat(data.seed), r2s = sqrt(r2);
	float3 w = nl;
	float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0.0f, 1.0f, 0.0f) : make_float3(1.0f, 0.0f, 0.0f)), w));
	float3 v = cross(w, u);
	float3 d = normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrt(1 - r2));

	// Possibly optional line?
//	if (++depth>5) if (randFloat()<p) f = f*(1 / p); else return obj.e;
	float3 direct = make_float3(0.0f, 0.0f, 0.0f);
	for (int i = 0; i < lights.size(); i++) {
		const SphereLight& s = lights[i];
		float3 sw = s.p - pos, su = normalize(cross((fabs(sw.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), sw)), sv = cross(sw, su);
		float cos_a_max = sqrt(1 - s.r*s.r / dot(pos - s.p, pos - s.p));
		float eps1 = randFloat(data.seed), eps2 = randFloat(data.seed);
		float cos_a = 1 - eps1 + eps2*cos_a_max;
		float sin_a = sqrt(1 - cos_a*cos_a);
		float phi = 2 * M_PI * eps2;
		float3 l = normalize(su*cos(phi)*sin_a + sv*sin(phi)*sin_a + sw*cos_a);
		Ray ray = make_Ray(pos, l, 0, DEFAULT_MIN, DEFAULT_MAX);
		RayData payload;
		payload.id = s.id - 1;	// TODO: update to shadow ray to avoid shading
		rtTrace(objects, ray, payload);
		if (payload.id == s.id) {
			float omega = 2 * M_PI * (1 - cos_a_max);
			direct += f * (s.e * dot(l, n) * omega) * M_1_PIf;
		}
	}
	
	data.result = e*data.E + direct + (f * radiance(pos, d, data.depth + 1, 0));
}

RT_PROGRAM void specular() {	// Ideal specular reflection
	data.id = id;
	if (data.depth > maxDepth) {
		data.result = e;
		return;
	}
	float3 pos = r.origin + r.direction*t;
	float3 nl = dot(n, r.direction) < 0 ? n : -n;
	float p = fmaxf(f); // max reflectance

	// Possibly optional line?
	//	if (++depth>5) if (randFloat()<p) f = f*(1 / p); else return obj.e;

	data.result = e + (f * radiance(pos, r.direction - (n * 2 * dot(n, r.direction)), data.depth + 1));
}

RT_PROGRAM void refractive() {	// Ideal dielectric REFRACTION
	data.id = id;
	if (data.depth > maxDepth) {
		data.result = e;
		return;
	}
	float3 pos = r.origin + r.direction*t;
	float3 nl = dot(n, r.direction) < 0 ? n : -n;
	float p = fmaxf(f); // max reflectance

	// Possibly optional line?
	//	if (++depth>5) if (randFloat()<p) f = f*(1 / p); else return obj.e;

	float3 reflDir = r.direction - (n * 2 * dot(n, r.direction));
	bool into = dot(n, nl) > 0;
	float nc = 1, nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = dot(r.direction, nl), cos2t;
	if ((cos2t = 1 - nnt*nnt*(1 - ddn*ddn)) < 0) {	// Total internal reflection
		data.result = e + (f * radiance(pos, reflDir, data.depth + 1));
		return;
	}
	float3 tdir = normalize(r.direction*nnt - n*((into ? 1 : -1)*(ddn*nnt + sqrt(cos2t))));
	float a = nt - nc, b = nt + nc, R0 = a*a / (b*b), c = 1 - (into ? -ddn : dot(tdir, n));
	float Re = R0 + (1 - R0)*c*c*c*c*c, Tr = 1 - Re, P = .25 + .5*Re, RP = Re / P, TP = Tr / (1 - P);
	data.result = e + (f * (data.depth > 2 ? (randFloat(data.seed) < P ?   // Russian roulette
		radiance(pos, reflDir, data.depth + 1)*RP : radiance(pos, tdir, data.depth + 1)*TP) :
		radiance(pos, reflDir, data.depth + 1)*Re + radiance(pos, tdir, data.depth + 1)*Tr));
}