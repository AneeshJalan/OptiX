#include "hip/hip_runtime.h"
#include "RayData.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

rtBuffer<uchar4, 2> output;
rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtDeclareVariable(uint2, dim, rtLaunchDim, );
rtDeclareVariable(rtObject, objects, , );
rtDeclareVariable(int, samples, , );
rtDeclareVariable(float3, camOrigin, , );
rtDeclareVariable(float3, camDirection, , );

static __device__ __inline__ float3 radiance(float3 o, float3 d, int depth, unsigned& seed) {
	//return make_float3(randFloat(seed), randFloat(seed), randFloat(seed));
	RayData payload;
	payload.E = 1;
	payload.depth = depth;
	payload.seed = seed;
	optix::Ray ray = optix::make_Ray(o, d, 0, DEFAULT_MIN, DEFAULT_MAX);
	rtTrace(objects, ray, payload);
	return payload.result;
}

static __device__ __inline__ unsigned char correct(float color) {
	return static_cast<unsigned char>(powf(clamp(color, 0.0f, 1.0f), 1 / 2.2) * 255 + .5);
}

static __device__ __inline__ uchar4 convertColor(float3 clr) {
	return make_uchar4(correct(clr.z), correct(clr.y), correct(clr.x), 255u);
}

RT_PROGRAM void trace() {
	float3 cx = make_float3(dim.x*0.5135f / dim.y, 0.0f, 0.0f);
	float3 cy = normalize(cross(cx, camDirection)) * 0.5135f;
	unsigned seed = createSeed(dim.x*index.y + index.x, dim.y*index.x + index.y);
	float3 result = make_float3(0.0f, 0.0f, 0.0f);
	for (int sy = 0; sy < 2; sy++) {		// 2x2 subpixel rows
		for (int sx = 0; sx < 2; sx++) {				// 2x2 subpixel cols
			float3 r = make_float3(0.0f, 0.0f, 0.0f);
			for (int s = 0; s < samples; s++) {
				float r1 = 2 * randFloat(seed), dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
				float r2 = 2 * randFloat(seed), dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);
				float3 d =	cx*(((sx + 0.5f + dx) / 2 + index.x) / dim.x - 0.5f) +
							cy*(((sy + 0.5f + dy) / 2 + index.y) / dim.y - 0.5f) + camDirection;
				r += radiance(camOrigin + d * 140, normalize(d), 0, seed);
			} // Camera rays are pushed ^^^^^ forward to start in interior
			result += clamp(r / samples, 0.0f, 1.0f);
		}
	}
	output[index] = convertColor(result * 0.25f);
}

RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, index.x, index.y);
	output[index] = convertColor(make_float3(1.0f, 0.0f, 1.0f));
}

rtDeclareVariable(RayData, data, rtPayload, );

RT_PROGRAM void miss() {
	data.result = make_float3(1.0f, 0.0f, 0.0f);
}