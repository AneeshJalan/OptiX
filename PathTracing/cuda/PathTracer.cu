#include "hip/hip_runtime.h"
#include "Utilities.h"
#include "SphereLight.h"

#define T_MIN 1e-4
#define T_MAX 1e+4

struct RayData {
	float3 result;
	unsigned depth;
	unsigned* seed;
};

rtBuffer<float4, 2> image;
rtDeclareVariable(uint2, dim, rtLaunchDim, );
rtDeclareVariable(uint2, pixel, rtLaunchIndex, );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, objects, , );
rtDeclareVariable(int, NUM_SAMPLES, , );
rtDeclareVariable(int, MAX_DEPTH, , );
rtDeclareVariable(int, frame, , );

rtBuffer<SphereLight> lights;
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RayData, current_prd, rtPayload, );
rtDeclareVariable(Ray, current_ray, rtCurrentRay, );
rtDeclareVariable(float3, n, attribute normal, );
rtDeclareVariable(float3, e, attribute emission, );
rtDeclareVariable(float3, f, attribute color, );

RT_PROGRAM void pathTrace() {
	uint2 dim = image.size();
	float2 invDim2 = 2.0 / make_float2(dim);
	float3 acc = make_float3(0.0f, 0.0f, 0.0f);
	for (int sample = 0; sample < NUM_SAMPLES; sample++) {
		unsigned seed = createSeed(pixel.y*pixel.y*pixel.x, frame * NUM_SAMPLES + sample);
		float2 point = (make_float2(pixel) + make_float2(rand(seed), rand(seed))) * invDim2 - 1.0f;	// Random pixel in scene
		float3 dir = normalize(point.x * U + point.y * V + W);
		Ray ray = make_Ray(eye, dir, 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = 0;
		new_prd.seed = &seed;
		rtTrace(objects, ray, new_prd);
		acc += new_prd.result;
	}
	acc *= 1.0f / float(NUM_SAMPLES);
	if (frame == 0) image[pixel] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	image[pixel] += make_float4(acc, 1.0f);
}

RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d, %d)\n", code, pixel.x, pixel.y);
	image[pixel] = make_float4(1.0f, 0.0f, 1.0f, 1.0f);
}

RT_PROGRAM void miss() {
	current_prd.result = make_float3(0.0f, 0.0f, 0.0f);
}

#define DL_CHANCE 0.2f

RT_PROGRAM void diffuse() {
	if (current_prd.depth < MAX_DEPTH && length(e) < 1) {
		float3 pos = current_ray.origin + t * current_ray.direction;
		/*/ TODO: Direct component
		float3 dl = make_float3(0.0f, 0.0f, 0.0f);
		for (int i = 0; i < lights.size(); i++) {
			float3 ln = pos - lights[i].position;
			float3 sample = lights[i].position + lights[i].radius * cosineSample(normalize(ln), *current_prd.seed);
			float3 dir = sample - pos;
			float ndl = dot(n, dir);
			if (ndl > 0) {
				Ray ray = make_Ray(pos, normalize(dir), 0, T_MIN, length(ln));
				RayData new_prd;
				new_prd.depth = current_prd.depth + 1;
				new_prd.seed = current_prd.seed;
				rtTrace(objects, ray, new_prd);
				dl += new_prd.result * ndl / M_PI;
			}
		}	*/
		// Indirect component:
		// Pick a random direction from here and keep going.

		//Randomized direct component
		float3 dir = cosineSample(n, *current_prd.seed);
		float weight = 1.0;
		float r = rand(*current_prd.seed);
		if (r < DL_CHANCE) {
			int i = (int)(lights.size() * r / DL_CHANCE);
			float3 ln = pos - lights[i].position;
			float3 point = normalize(cosineSample(normalize(ln), *current_prd.seed));
			float3 sample = lights[i].position + lights[i].radius * point;
			float ndl = dot(n, sample - pos);
			if (ndl > 0) {
				dir = sample - pos;
				weight = ndl * M_1_PIf * 0.5f * (lights[i].radius * lights[i].radius) / dot(ln, ln);	//proportional area of  light over hemisphere * BRDF
			}
		}	

		Ray ray = make_Ray(pos, normalize(dir), 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = current_prd.depth + 1;
		new_prd.seed = current_prd.seed;
		rtTrace(objects, ray, new_prd);

		// Apply the Rendering Equation here.
		current_prd.result = e + f * (weight * new_prd.result);
		return;
	}
	current_prd.result = e;
}