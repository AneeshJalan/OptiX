#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

#define T_MIN 1e-4
#define T_MAX 1e+4
#define M_PI 3.141592653589793

using namespace optix;

struct __align__(16) RayData {
	float3 result;
	unsigned depth;
	unsigned* seed;
};

rtBuffer<float4, 2> image;
rtDeclareVariable(uint2, dim, rtLaunchDim, );
rtDeclareVariable(uint2, pixel, rtLaunchIndex, );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, objects, , );
rtDeclareVariable(int, NUM_SAMPLES, , );
rtDeclareVariable(int, MAX_DEPTH, , );

rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RayData, current_prd, rtPayload, );
rtDeclareVariable(Ray, current_ray, rtCurrentRay, );
rtDeclareVariable(float3, n, attribute normal, );
rtDeclareVariable(float3, e, attribute emission, );
rtDeclareVariable(float3, f, attribute color, );


// Initialize random seed
static __device__ __inline__ unsigned createSeed(unsigned v0, unsigned v1) {
	for (unsigned n = 0, s0 = 0; n < 16; n++) {
		s0 += 0x9e3779b9;
		v0 += ((v1 << 4) + 0xa341316c) ^ (v1 + s0) ^ ((v1 >> 5) + 0xc8013ea4);
		v1 += ((v0 << 4) + 0xad90777d) ^ (v0 + s0) ^ ((v0 >> 5) + 0x7e95761e);
	}
	return v0;
}

// LCG normalize from 0 to 1
static __device__ __inline__ float rand(unsigned& seed) {
	const unsigned int LCG_A = 1664525u;
	const unsigned int LCG_C = 1013904223u;
	seed = (LCG_A * seed + LCG_C);
	return float(seed & 0x00FFFFFF) / float(0x01000000);
}

RT_PROGRAM void pathTrace() {
	uint2 dim = image.size();
	float2 invDim2 = 2.0 / make_float2(dim);
	unsigned seed = createSeed(pixel.y*pixel.y*pixel.x, pixel.x*pixel.x*pixel.y);	// TODO: optimize seed/lcg
	float3 acc = make_float3(0.0f, 0.0f, 0.0f);
	for (int sample = 0; sample < NUM_SAMPLES; sample++) {
		float2 point = (make_float2(pixel) + make_float2(rand(seed), rand(seed))) * invDim2 - 1.0f;	// Random pixel in scene
		float3 dir = normalize(point.x * U + point.y * V + W);
		Ray ray = make_Ray(eye, dir, 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = 0;
		new_prd.seed = &seed;
		rtTrace(objects, ray, new_prd);
		acc += new_prd.result;
	}
	acc *= 1.0f / float(NUM_SAMPLES);
	image[pixel] = make_float4(clamp(acc, 0.0f, 1.0f), 1.0f);
}

RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d, %d)\n", code, pixel.x, pixel.y);
	image[pixel] = make_float4(1.0f, 0.0f, 1.0f, 1.0f);
}


RT_PROGRAM void miss() {
	current_prd.result = make_float3(0.0f, 0.0f, 0.0f);
}

RT_PROGRAM void diffuse() {
	if (current_prd.depth < MAX_DEPTH && length(e) < 1) {
		
		// Pick a random direction from here and keep going.
		float3 normal = n;
		float3 tangent = normalize(cross(n, make_float3(0, 1, n.z < 0 ? 1 : -1)));
		float3 binormal = normalize(cross(n, tangent));

		float theta = 2.0f * M_PI * rand(*current_prd.seed);
		float phi = 0.5f * M_PI *rand(*current_prd.seed);

		float3 pos = current_ray.origin + t * current_ray.direction;
		float3 dir = cosf(phi) * (cosf(theta) * tangent + sinf(theta) * binormal) + sinf(phi) * normal;

		Ray ray = make_Ray(pos, normalize(dir), 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = current_prd.depth + 1;
		new_prd.seed = current_prd.seed;

		// Compute the BRDF for this ray (assuming Lambertian reflection)
		float cos_theta = dot(dir, normal);
		float3 BRDF = 2 * f * cos_theta;
		rtTrace(objects, ray, new_prd);

		// Apply the Rendering Equation here.
		current_prd.result = e + (BRDF * new_prd.result);
		return;
	}
	current_prd.result = e;
}