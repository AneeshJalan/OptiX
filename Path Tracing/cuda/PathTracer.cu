#include "hip/hip_runtime.h"
#include "Utilities.h"
#include "SphereLight.h"

#define T_MIN 1e-4
#define T_MAX 1e+4
#define DL_CHANCE 0.5f

struct RayData {
	float3 result;
	unsigned depth;
	unsigned* seed;
	int id;
};

// Buffers to store different components of the image
rtBuffer<float3, 2> accum;	// Accumulation buffer for color
rtBuffer<uchar4, 2> image;	// Buffer for image, set to accum / frame, w is object id

// Variables describing the viewport and camera
rtDeclareVariable(uint2, dim, rtLaunchDim, );
rtDeclareVariable(uint2, pixel, rtLaunchIndex, );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

// Variables describing the scene
rtBuffer<PotentialLight> lights;
rtDeclareVariable(rtObject, objects, , );
rtDeclareVariable(int, samples, , );
rtDeclareVariable(int, maxDepth, , );
rtDeclareVariable(int, frame, , );

RT_PROGRAM void pathTrace() {
	float2 invDim2 = 2.0 / make_float2(dim);
	float3 acc = make_float3(0.0f, 0.0f, 0.0f);
	unsigned seed = createSeed(pixel.y*dim.x + pixel.x, frame);
	RayData new_prd;
	new_prd.seed = &seed;
	Ray ray = make_Ray(eye, acc, 0, T_MIN, T_MAX);	// acc is 0,0,0
	for (int sample = 0; sample < samples; sample++) {
		float2 point = (make_float2(pixel) + make_float2(rand(seed), rand(seed))) * invDim2 - 1.0f;	// Random pixel in scene
		ray.direction = normalize(point.x * U + point.y * V + W);
		new_prd.depth = 0;
		rtTrace(objects, ray, new_prd);
		float luma = luminance(new_prd.result);// *0.5f;
		acc += new_prd.result / (1.0f + luma);
	}
	accum[pixel] = (frame <= 1) ? acc / float(samples) : accum[pixel] + acc / float(samples);
	float3 result = correct(clamp(accum[pixel] / float(frame), 0.0f, 1.0f));
	image[pixel] = make_uchar4( (unsigned char)(result.x * 255 + 0.5f),
								(unsigned char)(result.y * 255 + 0.5f),
								(unsigned char)(result.z * 255 + 0.5f),
								(unsigned char)new_prd.id);
}

RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d, %d)\n", code, pixel.x, pixel.y);
	image[pixel] = make_uchar4(255, 0, 255, 0);
}

// Variables describing the current ray intersection
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RayData, current_prd, rtPayload, );
rtDeclareVariable(Ray, current_ray, rtCurrentRay, );
rtDeclareVariable(float3, n, attribute normal, );
rtDeclareVariable(float3, e, attribute emission, );
rtDeclareVariable(float3, f, attribute color, );
rtDeclareVariable(int, id, attribute ID, );

RT_PROGRAM void miss() {
	current_prd.result = make_float3(0.0f, 0.0f, 0.0f);
	current_prd.id = 0;
}

RT_PROGRAM void diffuse() {
	if (current_prd.depth < maxDepth && length(e) < 1) {
		float3 pos = current_ray.origin + t * current_ray.direction;
		float3 dir = cosineSample(n, *current_prd.seed);

		//Randomized direct component
		float weight = 1.0;
		float r = rand(*current_prd.seed);
		if (r < DL_CHANCE) {
			int i = (int)(lights.size() * r / DL_CHANCE);
			float3 ln = pos - lights[i].position;
			float3 point = normalize(cosineSample(normalize(ln), *current_prd.seed));
			float3 sample = lights[i].position + lights[i].radius * point;
			float ndl = dot(n, sample - pos);
			if (ndl > 0) {
				dir = sample - pos;
				// Weight proportionally to area of  light over hemisphere * BRDF
				weight = 0.5f * (lights[i].radius * lights[i].radius) / dot(ln, ln) / DL_CHANCE;	
			}
		}	

		// Recurse
		Ray ray = make_Ray(pos, normalize(dir), 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = current_prd.depth + 1;
		new_prd.seed = current_prd.seed;
		rtTrace(objects, ray, new_prd);

		current_prd.result = e + f * (weight * new_prd.result);
		if (current_prd.depth == 0) current_prd.id = id;
		return;
	}
	current_prd.result = e;
	current_prd.id = id;
}

RT_PROGRAM void reflect() {
	if (current_prd.depth < maxDepth && length(e) < 1) {
		float3 pos = current_ray.origin + t * current_ray.direction;
		float3 dir = reflect(current_ray.direction, n);

		Ray ray = make_Ray(pos, normalize(dir), 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = current_prd.depth + 1;
		new_prd.seed = current_prd.seed;
		rtTrace(objects, ray, new_prd);

		// Apply the Rendering Equation here.
		current_prd.result = e + (f * new_prd.result);
		if (current_prd.depth == 0) current_prd.id = id;
		return;
	}
	current_prd.result = e;
}

RT_PROGRAM void refract() {
	if (current_prd.depth < maxDepth && length(e) < 1) {
		float3 pos = current_ray.origin + t * current_ray.direction;
		float3 dir;
		if (!refract(dir, current_ray.direction, n, 1.5f)) {
			dir = reflect(current_ray.direction, -n);
		}

		Ray ray = make_Ray(pos, normalize(dir), 0, T_MIN, T_MAX);
		RayData new_prd;
		new_prd.depth = current_prd.depth + 1;
		new_prd.seed = current_prd.seed;
		rtTrace(objects, ray, new_prd);

		// Apply the Rendering Equation here.
		current_prd.result = e + (f * new_prd.result);
		if (current_prd.depth == 0) current_prd.id = id;
		return;
	}
	current_prd.result = e;
	current_prd.id = id;
}