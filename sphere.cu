#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float4, sphere, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx) {
	float3 O = ray.origin;
	float3 D = ray.direction;
	float3 C = make_float3(sphere);

	float a = dot(D, D);
	float b = 2.0f * dot(D, O - C);
	float c = dot(O - C, O - C) - (sphere.w * sphere.w);

	float d = (b*b) - (4.0f*a*c);
	if (d >= 0) {
		float result = ray.tmax;
		d = sqrtf(d);
		a *= 2;
		float t = (-b + d) / a;
		if (t > 1.0e-4f) result = t;
		t = (-b - d) / a;
		if (t > 1.0e-4f) result = fminf(result, t);
		if (rtPotentialIntersection(result)) {
			float3 pos = O + result*D;
			shading_normal = geometric_normal = pos - C;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	if (sphere.w > 0) {
		float3 C = make_float3(sphere);
		float3 R = make_float3(sphere.w);
		aabb->m_max = C + R;
		aabb->m_min = C - R;
	} else {
		aabb->invalidate();
	}
}